#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <typeinfo>
#include <vector>
#include <assert.h>

#define STAGGERED_TRANSFER
#define TRANSFER_DEBUG

namespace quda {

  using namespace quda::colorspinor;

  template<typename real, int nSpin, int nColor, int nVec, QudaFieldOrder order>
  struct FillVArg {

    FieldOrderCB<real,nSpin,nColor,nVec,order> V;
    FieldOrderCB<real,nSpin,nColor,1,order> B;
    const int v;

    FillVArg(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int v)
      : V(V), B(*(B[v])), v(v) { }

  };

  // CPU routine to copy the null-space vectors into the V-field
  template <typename Float, int nSpin, int nColor, int nVec, typename Arg>
  void FillVCPU(Arg &arg, int v) {

    for (int parity=0; parity<arg.V.Nparity(); parity++) {
      for (int x_cb=0; x_cb<arg.V.VolumeCB(); x_cb++) {
	for (int s=0; s<nSpin; s++) {
	  for (int c=0; c<nColor; c++) {
	    arg.V(parity, x_cb, s, c, arg.v) = arg.B(parity, x_cb, s, c);
	  }
	}
      }
    }

  }

  // GPU kernel to copy the null-space vectors into the V-field
  template <typename Float, int nSpin, int nColor, int nVec, typename Arg>
  __global__ void FillVGPU(Arg arg, int v) {

    int x_cb = threadIdx.x + blockDim.x*blockIdx.x;
    int parity = threadIdx.y + blockDim.y*blockIdx.y;

    for (int s=0; s<nSpin; s++) {
      for (int c=0; c<nColor; c++) {
	arg.V(parity, x_cb, s, c, arg.v) = arg.B(parity, x_cb, s, c);
      }
    }

  }

  template <typename real, int nSpin, int nColor, int nVec>
  class FillVLaunch : public TunableVectorY {

    ColorSpinorField &V;
    const std::vector<ColorSpinorField*> &B;
    const int v;
    unsigned int minThreads() const { return V.VolumeCB(); }

  public:
    FillVLaunch(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, const int v)
      : TunableVectorY(2), V(V), B(B), v(v) {
      (V.Location() == QUDA_CPU_FIELD_LOCATION) ? strcpy(aux,"CPU") : strcpy(aux,"GPU");
    }
    virtual ~FillVLaunch() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
	if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
	  FillVArg<real,nSpin,nColor,nVec,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER> arg(V,B,v);
	  FillVCPU<real,nSpin,nColor,nVec>(arg,v);
	} else {
	  errorQuda("Field order not implemented %d", V.FieldOrder());
	}
      } else {
	if (V.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
	  FillVArg<real,nSpin,nColor,nVec,QUDA_FLOAT2_FIELD_ORDER> arg(V,B,v);
	  FillVGPU<real,nSpin,nColor,nVec> <<<tp.grid,tp.block,tp.shared_bytes>>>(arg,v);
	} else {
	  errorQuda("Field order not implemented %d", V.FieldOrder());
	}
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(V.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; }
    long long bytes() const { return 2*V.Bytes(); }
  };


  template <typename real, int nSpin, int nColor, int nVec>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B) {
    for (int v=0; v<nVec; v++) {
      FillVLaunch<real,nSpin,nColor,nVec> f(V,B,v);
      f.apply(0);
    }
  }

  // For staggered this does not include factor 2 due to parity decomposition!
  template <typename Float, int nSpin, int nColor>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (Nvec == 2) {
      FillV<Float,nSpin,nColor,2>(V,B);
    } else if (Nvec == 4) {
      FillV<Float,nSpin,nColor,4>(V,B);
    } else if (Nvec == 8) {
      FillV<Float,nSpin,nColor,8>(V,B);
    } else if (Nvec == 12) {
      FillV<Float,nSpin,nColor,12>(V,B);
    } else if (Nvec == 16) {
      FillV<Float,nSpin,nColor,16>(V,B);
    } else if (Nvec == 20) {
      FillV<Float,nSpin,nColor,20>(V,B);
    } else if (Nvec == 24) {
      FillV<Float,nSpin,nColor,24>(V,B);
#define TRANSFER_DEBUG
    } else if (Nvec == 32) {
      FillV<Float,nSpin,nColor,32>(V,B);
    } else if (Nvec == 48) {
      FillV<Float,nSpin,nColor,48>(V,B);
#endif
    } else {
      errorQuda("Unsupported Nvec %d", Nvec);
    }
  }

  template <typename Float, int nSpin>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (B[0]->Ncolor()*Nvec != V.Ncolor()) errorQuda("Something wrong here");

    if (B[0]->Ncolor() == 2) {
      FillV<Float,nSpin,2>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 3) {
      FillV<Float,nSpin,3>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 8) {
      FillV<Float,nSpin,8>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 16) {
      FillV<Float,nSpin,16>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 24) {
      FillV<Float,nSpin,24>(V,B,Nvec);
    } else if(B[0]->Ncolor() == 32) {
      FillV<Float,nSpin,32>(V,B,Nvec);
    } else {
      errorQuda("Unsupported nColor %d", B[0]->Ncolor());
    }
  }

  template <typename Float>
  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Nspin() == 4) {
      FillV<Float,4>(V,B,Nvec);
    } else if (V.Nspin() == 2) {
      FillV<Float,2>(V,B,Nvec);
#ifdef GPU_STAGGERED_DIRAC
    } else if (V.Nspin() == 1) {
      FillV<Float,1>(V,B,Nvec);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", V.Nspin());
    }
  }

  void FillV(ColorSpinorField &V, const std::vector<ColorSpinorField*> &B, int Nvec) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      FillV<double>(V,B,Nvec);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      FillV<float>(V,B,Nvec);
    } else {
      errorQuda("Unsupported precision %d", V.Precision());
    }
  }

  // Creates a block-ordered version of a ColorSpinorField
  // N.B.: Only works for the V field, as we need to block spin.
  template <bool toBlock, int nVec, class Complex, class FieldOrder>
  void blockOrderV(Complex *out, FieldOrder &in,
		   const int *geo_map, const int *geo_bs, int spin_bs,
		   const cpuColorSpinorField &V) {
    //printfQuda("in.Ncolor = %d\n", in.Ncolor());
    int nSpin_coarse = in.Nspin() / spin_bs; // this is number of chiral blocks

    //Compute the size of each block
    int geoBlockSize = 1;
    for (int d=0; d<in.Ndim(); d++) geoBlockSize *= geo_bs[d];
    int blockSize = geoBlockSize * in.Ncolor() * spin_bs; // blockSize includes internal dof

    int x[QUDA_MAX_DIM]; // global coordinates
    int y[QUDA_MAX_DIM]; // local coordinates within a block (full site ordering)

    int checkLength = in.Nparity() * in.VolumeCB() * in.Ncolor() * in.Nspin() * in.Nvec();
    int *check = new int[checkLength];
    int count = 0;

    // Run through the fine grid and do the block ordering
    for (int parity = 0; parity<in.Nparity(); parity++) {
      for (int x_cb=0; x_cb<in.VolumeCB(); x_cb++) {
	int i = parity*in.VolumeCB() + x_cb;

	// Get fine grid coordinates
	V.LatticeIndex(x, i);
	
	//Compute the geometric offset within a block 
	// (x fastest direction, t is slowest direction, non-parity ordered)
	int blockOffset = 0;
	for (int d=in.Ndim()-1; d>=0; d--) {
	  y[d] = x[d]%geo_bs[d];
	  blockOffset *= geo_bs[d];
	  blockOffset += y[d];
	}
	
	//Take the block-ordered offset from the coarse grid offset (geo_map) 
	int offset = geo_map[i]*nSpin_coarse*nVec*geoBlockSize*in.Ncolor()*spin_bs;
	
	for (int v=0; v<in.Nvec(); v++) {
	  for (int s=0; s<in.Nspin(); s++) {
	    for (int c=0; c<in.Ncolor(); c++) {
	      
	      int chirality = s / spin_bs; // chirality is the coarse spin
	      int blockSpin = s % spin_bs; // the remaining spin dof left in each block
	      
	      int index = offset +                                              // geo block
		chirality * nVec * geoBlockSize * spin_bs * in.Ncolor() + // chiral block
	                       v * geoBlockSize * spin_bs * in.Ncolor() + // vector
	                            blockOffset * spin_bs * in.Ncolor() + // local geometry
	                                          blockSpin*in.Ncolor() + // block spin
	                                                                   c;   // color
#ifndef STAGGERED_TRANSFER
	      if (toBlock) out[index] = in(parity, x_cb, s, c, v); // going to block order
#else
	      if (toBlock) out[index] = parity == 0 ? in(parity, x_cb, s, c, v) : 0.0; // going to block order
#endif
	      else in(parity, x_cb, s, c, v) = out[index]; // coming from block order
	    
	      check[count++] = index;
	    }
	  }
	}
      }

      //printf("blockOrderV done %d / %d\n", i, in.Volume());
    }
    
    if (count != checkLength) {
      errorQuda("Number of elements packed %d does not match expected value %d nvec=%d nspin=%d ncolor=%d", 
		count, checkLength, in.Nvec(), in.Nspin(), in.Ncolor());
    }

    /*
    // need non-quadratic check
    for (int i=0; i<checkLength; i++) {
      for (int j=0; j<i; j++) {
      if (check[i] == check[j]) errorQuda("Collision detected in block ordering\n");
      }
    }
    */
    delete []check;
  }


  // Orthogonalise the nc vectors v[] of length n
  // this assumes the ordering v[(b * Nvec + v) * blocksize + i]

  template <typename sumFloat, typename Float, int N>
  void blockGramSchmidt(complex<Float> *v, int nBlocks, int blockSize) {
    
    for (int b=0; b<nBlocks; b++) {
      for (int jc=0; jc<N; jc++) {
      
	for (int ic=0; ic<jc; ic++) {
	  // Calculate dot product.
	  complex<Float> dot = 0.0;
	  for (int i=0; i<blockSize; i++) 
	    dot += conj(v[(b*N+ic)*blockSize+i]) * v[(b*N+jc)*blockSize+i];
	  
	  // Subtract the blocks to orthogonalise
	  for (int i=0; i<blockSize; i++) 
	    v[(b*N+jc)*blockSize+i] -= dot * v[(b*N+ic)*blockSize+i];
	}
	
	// Normalize the block
	// nrm2 is pure real, but need to use Complex because of template.
        sumFloat nrm2 = 0.0;
	for (int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	sumFloat scale = nrm2 > 0.0 ? 1.0/sqrt(nrm2) : 0.0;
	for (int i=0; i<blockSize; i++) v[(b*N+jc)*blockSize+i] *= scale;
      }

      /*      
      for (int jc=0; jc<N; jc++) {
        complex<sumFloat> nrm2 = 0.0;
        for(int i=0; i<blockSize; i++) nrm2 += norm(v[(b*N+jc)*blockSize+i]);
	//printfQuda("block = %d jc = %d nrm2 = %f\n", b, jc, nrm2.real());
      }
      */

      //printf("blockGramSchmidt done %d / %d\n", b, nBlocks);
    }

  }

  template <typename sumType, typename real, int N>
  class BlockGramSchmidt : public Tunable {

    complex<real> *v;
    int nBlock;
    int blockSize;
    const ColorSpinorField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  public:
    BlockGramSchmidt(complex<real> *v, int nBlock, int blockSize, const ColorSpinorField &meta)
      : v(v), nBlock(nBlock), blockSize(blockSize), meta(meta) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) sprintf(aux, "nBlock=%d,blockSize=%d,CPU", nBlock, blockSize);
      else sprintf(aux, "nBlock=%d,blockSize=%d,GPU", nBlock, blockSize);
    }

    virtual ~BlockGramSchmidt() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
	blockGramSchmidt<sumType, real, N>(v, nBlock, blockSize);
      } else {
	errorQuda("Not implemented for GPU");
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return nBlock * N * ((N-1) * (8l + 8l) + 2l) * blockSize; }
    long long bytes() const { return 2*meta.Bytes(); }
  };

  template <bool toBlock, int N, typename real, typename Order>
  class BlockOrderV : public Tunable {

    complex<real> *vBlock;
    Order &vOrder;
    const int *geo_map;
    const int *geo_bs;
    int spin_bs;
    const ColorSpinorField &V;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

  public:
    BlockOrderV(complex<real> *vBlock, Order &vOrder, const int *geo_map, const int *geo_bs, int spin_bs, const ColorSpinorField &V)
      : vBlock(vBlock), vOrder(vOrder), geo_map(geo_map), geo_bs(geo_bs), spin_bs(spin_bs), V(V) {
      (V.Location() == QUDA_CPU_FIELD_LOCATION) ? strcpy(aux, "CPU") : strcpy(aux,"GPU");
    }

    virtual ~BlockOrderV() { }

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (V.Location() == QUDA_CPU_FIELD_LOCATION) {
	blockOrderV<toBlock,N,complex<real>,Order>(vBlock,vOrder,geo_map,geo_bs,spin_bs,V);
      } else {
	errorQuda("Not implemented for GPU");
      }
    }

    bool advanceTuneParam(TuneParam &param) const { return false; }

    TuneKey tuneKey() const { return TuneKey(V.VolString(), typeid(*this).name(), aux); }

    long long flops() const { return 0; }
    long long bytes() const { return 2*V.Bytes(); }
  };

#if 0
  using namespace quda::colorspinor;

  /**
      Kernel argument struct
  */
  template <typename Out, typename In, typename Rotator, int fineSpin, int coarseSpin>
  struct BlockOrthoArg {
    const Rotator V;
    const int *fine_to_coarse;
    const int *coarse_to_fine;
    const spin_mapper<fineSpin,coarseSpin> spin_map;
    const int parity; // the parity of the input field (if single parity)
    const int nParity; // number of parities of input fine field
    int swizzle; // swizzle factor for transposing blockIdx.x mapping to coarse grid coordinate

    BlockOrthoArg(Rotator &V, const int *fine_to_coarse, const int *coarse_to_fine,
		  int parity, const ColorSpinorField &meta) :
      out(out), in(in), V(V), fine_to_coarse(fine_to_coarse), coarse_to_fine(coarse_to_fine),
      spin_map(), parity(parity), nParity(meta.SiteSubset()), swizzle(1)
    { }

    BlockOrthoArg(const BlockOrthoArg<Out,In,Rotator,fineSpin,coarseSpin> &arg) :
      out(arg.out), in(arg.in), V(arg.V),
      fine_to_coarse(arg.fine_to_coarse), coarse_to_fine(arg.coarse_to_fine), spin_map(),
      parity(arg.parity), nParity(arg.nParity), swizzle(arg.swizzle)
    { }
  };

  template <typename Float, int nVec, int fineSpin, int coarseSpin, typename Arg>
  void BlockOrtho(Arg &arg) {

    constexpr spinBlocks = fineSpin / coarseSpin;

    for (int b=0; b<nBlocks; b++) {
      for (int s=0; s<spinBlocks; s++) {

	for (int k=0; k<nVec; k++) {

	  for (int l=0; l<k; l++) {
	    complex<Float> dot = 0.0;

	    for (int i=0; i<blockSize; i++) {

	      dot += conj(v(parity, x_cb, s, c, l)) * v(parity, x_cb, s, c, k);

	    }

	}

      }
    }

    for (int parity_coarse=0; parity_coarse<2; parity_coarse++)
      for (int x_coarse_cb=0; x_coarse_cb<arg.out.VolumeCB(); x_coarse_cb++)
	for (int s=0; s<coarseSpin; s++)
	  for (int c=0; c<coarseColor; c++)
	    arg.out(parity_coarse, x_coarse_cb, s, c) = 0.0;

    // loop over fine degrees of freedom
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb=0; x_cb<arg.in.VolumeCB(); x_cb++) {

	int x = parity*arg.in.VolumeCB() + x_cb;
	int x_coarse = arg.fine_to_coarse[x];
	int parity_coarse = (x_coarse >= arg.out.VolumeCB()) ? 1 : 0;
	int x_coarse_cb = x_coarse - parity_coarse*arg.out.VolumeCB();

	for (int coarse_color_block=0; coarse_color_block<coarseColor; coarse_color_block+=coarse_colors_per_thread) {
	  complex<Float> tmp[fineSpin*coarse_colors_per_thread];
	  rotateCoarseColor<Float,fineSpin,fineColor,coarseColor,coarse_colors_per_thread>
	    (tmp, arg.in, arg.V, parity, arg.nParity, x_cb, coarse_color_block);

	  for (int s=0; s<fineSpin; s++) {
	    for (int coarse_color_local=0; coarse_color_local<coarse_colors_per_thread; coarse_color_local++) {
	      int c = coarse_color_block + coarse_color_local;
	      arg.out(parity_coarse,x_coarse_cb,arg.spin_map(s),c) += tmp[s*coarse_colors_per_thread+coarse_color_local];
	    }
	  }

	}
      }
    }

  }
#endif //end

    template<typename Float, int nSpin, int nColor, int nVec>
  void BlockOrthogonalize(ColorSpinorField &V, const int *geo_bs, const int *geo_map, int spin_bs) {
    complex<Float> *Vblock = new complex<Float>[V.Volume()*V.Nspin()*V.Ncolor()];

    if (V.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
      constexpr QudaFieldOrder order = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

      typedef FieldOrderCB<Float,nSpin,nColor,nVec,order> VectorField;
      VectorField vOrder(const_cast<ColorSpinorField&>(V));

      int geo_blocksize = 1;
      for (int d = 0; d < V.Ndim(); d++) geo_blocksize *= geo_bs[d];

      int blocksize = geo_blocksize * vOrder.Ncolor() * spin_bs;
      int chiralBlocks = (V.Nspin() == 1) ? 1 : vOrder.Nspin() / spin_bs; //always 1 for parity staggered (!).
      int numblocks = (V.Volume()/geo_blocksize) * chiralBlocks;
      if (V.Nspin() == 1) blocksize /= chiralBlocks; //for staggered chiral block size is a parity block size
    
      printfQuda("Block Orthogonalizing %d blocks of %d length and width %d\n", numblocks, blocksize, nVec);

#if 0
      BlockOrthoArg<> arg(V);
      BlockOrtho ortho();
      otho.apply(0);
#endif

      BlockOrderV<true,nVec,Float,VectorField> reorder(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
      reorder.apply(0);

      BlockGramSchmidt<double,Float,nVec> ortho(Vblock, numblocks, blocksize, V);
      ortho.apply(0);

      BlockOrderV<false,nVec,Float,VectorField> reset(Vblock, vOrder, geo_map, geo_bs, spin_bs, V);
      reset.apply(0);

      delete []Vblock;

    } else {
      errorQuda("Unsupported field order %d\n", V.FieldOrder());
    }

  }

  template<typename Float, int nSpin, int nColor>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, const int *geo_bs, const int *geo_map, int spin_bs) {
    if (Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,nColor,2>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 4) {
      BlockOrthogonalize<Float,nSpin,nColor,4>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,nColor,8>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 12) {
      BlockOrthogonalize<Float,nSpin,nColor,12>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,nColor,16>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 20) {
      BlockOrthogonalize<Float,nSpin,nColor,20>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,nColor,24>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 32) {
      BlockOrthogonalize<Float,nSpin,nColor,32>(V, geo_bs, geo_map, spin_bs);
    } else if (Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,nColor,48>(V, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported nVec %d\n", Nvec);
    }
  }

  template<typename Float, int nSpin>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Ncolor()/Nvec == 3) {
      BlockOrthogonalize<Float,nSpin,3>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Ncolor()/Nvec == 2) {
      BlockOrthogonalize<Float,nSpin,2>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Ncolor()/Nvec == 8) {
      BlockOrthogonalize<Float,nSpin,8>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Ncolor()/Nvec == 16) {
      BlockOrthogonalize<Float,nSpin,16>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Ncolor()/Nvec == 24) {
      BlockOrthogonalize<Float,nSpin,24>(V, Nvec, geo_bs, geo_map, spin_bs);
#ifdef TRANSFER_DEBUG
    } else if (V.Ncolor()/Nvec == 32) {
      BlockOrthogonalize<Float,nSpin,32>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Ncolor()/Nvec == 48) {
      BlockOrthogonalize<Float,nSpin,48>(V, Nvec, geo_bs, geo_map, spin_bs); //for staggered, even-odd blocking presumed
#endif
    }  
    else {
      errorQuda("Unsupported nColor %d\n", V.Ncolor()/Nvec);
    }
  }

  template<typename Float>
  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Nspin() == 4) {
      BlockOrthogonalize<Float,4>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if(V.Nspin() ==2) {
      BlockOrthogonalize<Float,2>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else if (V.Nspin() == 1) {
      BlockOrthogonalize<Float,1>(V, Nvec, geo_bs, geo_map, 1);
    }
    else {
      errorQuda("Unsupported nSpin %d\n", V.Nspin());
    }
  }

  void BlockOrthogonalize(ColorSpinorField &V, int Nvec, 
			  const int *geo_bs, const int *geo_map, int spin_bs) {
    if (V.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      BlockOrthogonalize<double>(V, Nvec, geo_bs, geo_map, spin_bs);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (V.Precision() == QUDA_SINGLE_PRECISION) {
      BlockOrthogonalize<float>(V, Nvec, geo_bs, geo_map, spin_bs);
    } else {
      errorQuda("Unsupported precision %d\n", V.Precision());
    }
  }

} // namespace quda
